#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
 * Copyright 2016 Colin Heinzmann                                            *
 *                                                                           *
 * Licensed under the Apache License, Version 2.0 (the "License");           *
 * you may not use this file except in compliance with the License.          *
 * You may obtain a copy of the License at                                   *
 *                                                                           *
 *     http://www.apache.org/licenses/LICENSE-2.0                            *
 *                                                                           *
 * Unless required by applicable law or agreed to in writing, software       *
 * distributed under the License is distributed on an "AS IS" BASIS,         *
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  *
 * See the License for the specific language governing permissions and       *
 * limitations under the License.                                            *
 * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

#include <iostream>

using namespace std;

#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <math.h>

#include "cudadefines.hpp"

using namespace cv;
using namespace cv::gpu;

namespace br { namespace cuda { namespace affine {

    __device__ __forceinline__ uint8_t getPixelValueDevice(int row, int col, uint8_t* srcPtr, int rows, int cols) {
        return (srcPtr + row*cols)[col];
    }


    __device__ __forceinline__ uint8_t getBilinearPixelValueDevice(double row, double col, uint8_t* srcPtr, int rows, int cols) {
        // http://www.sci.utah.edu/~acoste/uou/Image/project3/ArthurCOSTE_Project3.pdf
        // Bilinear Transformation
        // f(Px, Py) = f(Q11)×(1−Rx)×(1−Sy)+f(Q21)×(Rx)×(1−Sy)+f(Q12)×(1−Rx)×(Sy)+f(Q22)×(Rx)×(Sy)

        int row1 = floor(row);
        int row2 = row1+1;

        int col1 = floor(col);
        int col2 = col1+1;

        double d_row = row - row1;
        double d_col = col - col1;

        int Q11 = getPixelValueDevice(row1, col1, srcPtr, rows, cols);
        int Q21 = getPixelValueDevice(row2, col1, srcPtr, rows, cols);
        int Q12 = getPixelValueDevice(row1, col2, srcPtr, rows, cols);
        int Q22 = getPixelValueDevice(row2, col2, srcPtr, rows, cols);

        double val = Q22*(d_row*d_col) + Q12*((1-d_row)*d_col) + Q21*(d_row*(1-d_col)) + Q11*((1-d_row)*(1-d_col));
        return ((uint8_t) round(val));
    }

    __device__ __forceinline__ uint8_t getDistancePixelValueDevice(double row, double col, uint8_t* srcPtr, int rows, int cols) {
        int row1 = floor(row);
        int row2 = row1+1;

        int col1 = floor(col);
        int col2 = col1+1;

        double m1 = row2 - row;
        double m12 = m1*m1;

        double m2 = col - col1;
        double m22 = m2*m2;

        double d1 = sqrt(m12 - 2*m1 + 1 + m22);
        double d2 = sqrt(m12 + m22);
        double d3 = sqrt(m12 - 2*m1 + 1 + m22 - 2*m2 + 1);
        double d4 = sqrt(m12 + m22 - 2*m2 + 1);
        double sum = d1 + d2 + d3 + d4;

        double w1 = d1/sum;
        double w2 = d2/sum;
        double w3 = d3/sum;
        double w4 = d4/sum;

        uint8_t v1 = getPixelValueDevice(row1, col1, srcPtr, rows, cols);
        uint8_t v2 = getPixelValueDevice(row2, col1, srcPtr, rows, cols);
        uint8_t v3 = getPixelValueDevice(row1, col2, srcPtr, rows, cols);
        uint8_t v4 = getPixelValueDevice(row2, col2, srcPtr, rows, cols);

        return round(w1*v1 + w2*v2 + w3*v3 + w4*v4);
    }

    /*
     * trans_inv          - A pointer to a one-dimensional representation of the inverse of the transform matrix 3x3
     * dst_row            - The destination row (mapping to this row)
     * dst_col            - The destination column (mapping to this column)
     * src_row            - The computed source pixel row (mapping from this row)
     * src_col            - The computed source pixel column (mapping from this col)
     */
    __device__ __forceinline__ void getSrcCoordDevice(double *trans_inv, int dst_row, int dst_col, double* src_row_pnt, double* src_col_pnt){
        *src_col_pnt = dst_col * trans_inv[0] + dst_row * trans_inv[3] + trans_inv[6];
        *src_row_pnt = dst_col * trans_inv[1] + dst_row * trans_inv[4] + trans_inv[7];
		}

    __global__ void bilinearKernel(uint8_t* srcPtr, uint8_t* dstPtr, int srcRows, int srcCols, int dstRows, int dstCols) {
      int dstRowInd = blockIdx.y*blockDim.y+threadIdx.y;
      int dstColInd = blockIdx.x*blockDim.x+threadIdx.x;
      int dstIndex = dstRowInd*dstCols+dstColInd;

      // destination boundary checking
      if (dstRowInd >= dstRows || dstColInd >= dstCols) {
        return;
      }

      // get the reference indices and relative amounts
      float exactSrcRowInd = (float)dstRowInd / (float)dstRows * (float)srcRows;
      int minSrcRowInd = (int)exactSrcRowInd;
      int maxSrcRowInd = minSrcRowInd+1;
      float relSrcRowInd = 1.-(exactSrcRowInd-(float)minSrcRowInd);

      // get the reference indices and relative amounts
      double exactSrcColInd = (double)dstColInd / (double)dstCols * (double)srcCols;
      int minSrcColInd = (int)exactSrcColInd;
      int maxSrcColInd = minSrcColInd+1;
      float relSrcColInd = 1.-(exactSrcColInd-(float)minSrcColInd);

      // perform boundary checking
      if (minSrcRowInd < 0 || maxSrcRowInd >= srcRows || minSrcColInd < 0 || maxSrcColInd >= srcCols) {
        dstPtr[dstIndex] = 0;
        return;
      }

      // get each of the pixel values
      float topLeft = srcPtr[minSrcRowInd*srcCols+minSrcColInd];
      float topRight = srcPtr[minSrcRowInd*srcCols+maxSrcColInd];
      float bottomLeft = srcPtr[maxSrcRowInd*srcCols+minSrcColInd];
      float bottomRight = srcPtr[maxSrcRowInd*srcCols+maxSrcColInd];

      float out = relSrcRowInd*relSrcColInd*topLeft + relSrcRowInd*(1.-relSrcColInd)*topRight + (1.-relSrcRowInd)*relSrcColInd*bottomLeft + (1.-relSrcRowInd)*(1.-relSrcColInd)*bottomRight;

      dstPtr[dstIndex] = (int)out;
    }

    __global__ void affineKernel(uint8_t* srcPtr, uint8_t* dstPtr, double* trans_inv, int src_rows, int src_cols, int dst_rows, int dst_cols){
        int dstRowInd = blockIdx.y*blockDim.y+threadIdx.y;
        int dstColInd = blockIdx.x*blockDim.x+threadIdx.x;
        int dstIndex = dstRowInd*dst_cols + dstColInd;

        double srcRowPnt;
        double srcColPnt;

        // don't do anything if the index is out of bounds
        if (dstRowInd >= dst_rows || dstColInd >= dst_cols) {
          return;
        }
        if (dstRowInd == 0 || dstRowInd == dst_rows-1 || dstColInd ==0 || dstColInd == dst_cols-1) {
          dstPtr[dstIndex] = 0;
          return;
        }

        getSrcCoordDevice(trans_inv, dstRowInd, dstColInd, &srcRowPnt, &srcColPnt);
        const uint8_t cval = getBilinearPixelValueDevice(srcRowPnt, srcColPnt, srcPtr, src_rows, src_cols); // Get initial pixel value

        dstPtr[dstIndex] = cval;
    }

    void resizeWrapper(void* srcPtr, void** dstPtr, int srcRows, int srcCols, int dstRows, int dstCols) {
      // perform bilinear filtering

      // allocate space for destination
      hipError_t err;
      CUDA_SAFE_MALLOC(dstPtr, dstRows*dstCols*sizeof(uint8_t), &err);

      // call the bilinear kernel function
      dim3 threadsPerBlock(32, 16);
      dim3 numBlocks(dstCols/threadsPerBlock.x + 1,
                     dstRows/threadsPerBlock.y + 1);

      bilinearKernel<<<numBlocks, threadsPerBlock>>>((uint8_t*)srcPtr, (uint8_t*)*dstPtr, srcRows, srcCols, dstRows, dstCols);
      CUDA_KERNEL_ERR_CHK(&err);

      CUDA_SAFE_FREE(srcPtr, &err);
    }

    void wrapper(void* srcPtr, void** dstPtr, Mat affineTransform, int src_rows, int src_cols, int dst_rows, int dst_cols) {
        hipError_t err;
        double* gpuInverse;

        dim3 threadsPerBlock(32, 16);
        dim3 numBlocks(dst_cols/threadsPerBlock.x + 1,
                       dst_rows/threadsPerBlock.y + 1);

        //************************************************************************
        // Input affine is a 2x3 Mat whose transpose is used in the computations
        // [x, y, 1] = [u, v, 1] [ a^T | [0 0 1]^T ]
        // See "Digital Image Warping" by George Wolburg (p. 50)
        //************************************************************************

        // get new transform elements
        double a11 = affineTransform.at<double>(0, 0);
        double a12 = affineTransform.at<double>(1, 0);
        double a21 = affineTransform.at<double>(0, 1);
        double a22 = affineTransform.at<double>(1, 1);
        double a31 = affineTransform.at<double>(0, 2);
        double a32 = affineTransform.at<double>(1, 2);

        // compute transform inverse
        double det = 1 / (a11*a22 - a21*a12);

        double affineInverse[9];
        affineInverse[0] = a22 * det;
        affineInverse[1] = -a12 * det;
        affineInverse[2] = 0;
        affineInverse[3] = -a21 * det;
        affineInverse[4] = a11 * det;
        affineInverse[5] = 0;
        affineInverse[6] = (a21*a32 - a31*a22) * det;
        affineInverse[7] = (a31*a12 - a11*a32) * det;
        affineInverse[8] = (a11*a22 - a21*a12) * det;

        CUDA_SAFE_MALLOC(dstPtr, dst_rows*dst_cols*sizeof(uint8_t), &err);
        CUDA_SAFE_MALLOC(&gpuInverse, 3*3*sizeof(double), &err);

        CUDA_SAFE_MEMCPY(gpuInverse, affineInverse, 9*sizeof(double), hipMemcpyHostToDevice, &err);

        affineKernel<<<numBlocks, threadsPerBlock>>>((uint8_t*)srcPtr, (uint8_t*)(*dstPtr), gpuInverse, src_rows, src_cols, dst_rows, dst_cols);
        CUDA_KERNEL_ERR_CHK(&err);

        CUDA_SAFE_FREE(srcPtr, &err);
        CUDA_SAFE_FREE(gpuInverse, &err);
    }
}}}
